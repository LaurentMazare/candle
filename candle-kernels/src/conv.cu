#include "hip/hip_runtime.h"
#include "cuda_utils.cuh"
#include<stdint.h>

template <typename T>
__device__ void conv1d(
    const size_t src_numel,
    const size_t l_out,
    const size_t stride, 
    const size_t *info,
    const T *src,
    const T *kernel,
    T *dst
) {
  // src: (b_size, c_in, l_in)
  // k: (c_out, c_in, k_size)
  const size_t *src_dims = info;
  const size_t *src_s = info + 3;
  const size_t *k_dims = info + 6;
  const size_t *k_s = info + 9;
  const size_t dst_i = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t k_size = k_dims[2];
  const size_t k_over_2 = k_size / 2;
  const size_t c_out = k_dims[0];
  const size_t c_in = src_dims[1];
  const size_t l_in = src_dims[2];

  // TODO
  const size_t b_idx = dst_i / (l_out * c_out);
  const size_t dst_c_idx = (dst_i / l_out) % c_out;
  const size_t dst_l = dst_i % l_out;

  const size_t src_idx0 = b_idx * src_s[0];
  T d = 0;
  for (size_t offset = 0; offset < k_size; ++offset) {
    const size_t src_l_plus = stride * dst_l + offset;
    if (k_over_2 <= src_l_plus && src_l_plus < l_in + k_over_2) {
      const size_t src_l = src_l_plus - k_over_2;
      for (size_t src_c_idx = 0; src_c_idx < c_in; ++src_c_idx) {
        const size_t src_idx = src_idx0 + src_c_idx * src_s[1] + src_l * src_s[2];
        const size_t k_idx = dst_c_idx * k_s[0] + src_c_idx * k_s[1] + offset * k_s[2];
        d += src[src_idx] * kernel[k_idx];
      }
    }
  }
  dst[dst_i] = d;
}


#define CONV1D_OP(TYPENAME, FN_NAME) \
extern "C" __global__ void FN_NAME(  \
    const size_t src_numel, \
    const size_t num_dims, \
    const size_t stride, \
    const size_t *info, \
    const TYPENAME *src, \
    const TYPENAME *kernel, \
    TYPENAME *dst \
) {  \
  conv1d(src_numel, num_dims, stride, info, src, kernel, dst); \
} \

#if __CUDA_ARCH__ >= 800
CONV1D_OP(__hip_bfloat16, conv1d_bf16)
#endif

#if __CUDA_ARCH__ >= 530
CONV1D_OP(__half, conv1d_f16)
#endif

CONV1D_OP(float, conv1d_f32)
CONV1D_OP(double, conv1d_f64)
CONV1D_OP(uint8_t, conv1d_u8)
CONV1D_OP(uint32_t, conv1d_u32)

