#include "hip/hip_runtime.h"
// WARNING: THIS IS ONLY VALID ASSUMING THAT inp IS CONTIGUOUS!
// TODO: proper error reporting when ids are larger than v_size.
#include "cuda_utils.cuh"
#include<stdint.h>

#define EMB_OP(TYPENAME, INDEX_TYPENAME, FN_NAME) \
extern "C" __global__ void FN_NAME(  \
    const size_t numel,  \
    const size_t num_dims, \
    const size_t *info, \
    const INDEX_TYPENAME *ids, \
    const TYPENAME *inp, \
    TYPENAME *out, \
    const size_t h_size, \
    const size_t v_size \
) {  \
    const size_t *dims = info; \
    const size_t *strides = info + num_dims; \
    if (is_contiguous(num_dims, dims, strides)) { \
        for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) { \
            memcpy(&out[i * h_size], &inp[ids[i] * h_size], h_size * sizeof(TYPENAME)); \
        } \
    } \
    else { \
        for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) { \
            unsigned strided_i = get_strided_index(i, num_dims, dims, strides); \
            memcpy(&out[i * h_size], &inp[ids[strided_i] * h_size], h_size * sizeof(TYPENAME)); \
        } \
    } \
} \

template<typename T, typename I>
__device__ void index_select(
    const size_t numel,
    const size_t num_dims,
    const size_t *info,
    const I *ids,
    const T *inp,
    T *out,
    const size_t left_size,
    const size_t dim_size,
    const size_t right_size
) {
    const size_t *dims = info;
    const size_t *strides = info + num_dims;
    if (is_contiguous(num_dims, dims, strides)) {
        for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) {
            for (unsigned int j = 0; j < left_size; ++j) {
                memcpy(&out[(i + j * numel) * right_size], &inp[(j * dim_size + ids[i]) * right_size], right_size * sizeof(T));
            }
        }
    }
    else {
        for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) {
            unsigned strided_i = get_strided_index(i, num_dims, dims, strides);
            for (unsigned int j = 0; j < left_size; ++j) {
                memcpy(&out[(i + j * numel) * right_size], &inp[(j * dim_size + ids[strided_i]) * right_size], right_size * sizeof(T));
            }
        }
    }
}

#define IS_OP(TYPENAME, INDEX_TYPENAME, FN_NAME) \
extern "C" __global__ void FN_NAME(  \
    const size_t numel,  \
    const size_t num_dims, \
    const size_t *info, \
    const INDEX_TYPENAME *ids, \
    const TYPENAME *inp, \
    TYPENAME *out, \
    const size_t left_size, \
    const size_t dim_size, \
    const size_t right_size \
) { index_select(numel, num_dims, info, ids, inp, out, left_size, dim_size, right_size); } \

template<typename T, typename I>
__device__ void gather(
    const size_t numel,
    const I *ids,
    const T *inp,
    T *out,
    const size_t left_size,
    const size_t src_dim_size,
    const size_t ids_dim_size,
    const size_t right_size
) {
    for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) {
        size_t post = i % right_size;
        size_t idx = ids[i];
        size_t pre = i / (right_size * ids_dim_size);
        size_t src_i = (pre * src_dim_size + idx) * right_size + post;
        out[i] = inp[src_i];
    }
}

#define GATHER_OP(TYPENAME, INDEX_TYPENAME, FN_NAME) \
extern "C" __global__ void FN_NAME(  \
    const size_t numel,  \
    const INDEX_TYPENAME *ids, \
    const TYPENAME *inp, \
    TYPENAME *out, \
    const size_t left_size, \
    const size_t src_dim_size, \
    const size_t ids_dim_size, \
    const size_t right_size \
) { gather(numel, ids, inp, out, left_size, src_dim_size, ids_dim_size, right_size); } \

#if __CUDA_ARCH__ >= 800
EMB_OP(__hip_bfloat16, uint32_t, emb_u32_bf16)
EMB_OP(__hip_bfloat16, uint8_t, emb_u8_bf16)
IS_OP(__hip_bfloat16, uint32_t, is_u32_bf16)
IS_OP(__hip_bfloat16, uint8_t, is_u8_bf16)
GATHER_OP(__hip_bfloat16, uint32_t, gather_u32_bf16)
GATHER_OP(__hip_bfloat16, uint8_t, gather_u8_bf16)
#endif

#if __CUDA_ARCH__ >= 530
EMB_OP(__half, uint32_t, emb_u32_f16)
EMB_OP(__half, uint8_t, emb_u8_f16)
IS_OP(__half, uint32_t, is_u32_f16)
IS_OP(__half, uint8_t, is_u8_f16)
GATHER_OP(__half, uint32_t, gather_u32_f16)
GATHER_OP(__half, uint8_t, gather_u8_f16)
#endif

EMB_OP(float, uint32_t, emb_u32_f32)
EMB_OP(double, uint32_t, emb_u32_f64)
EMB_OP(uint8_t, uint32_t, emb_u32_u8)
EMB_OP(uint32_t, uint32_t, emb_u32_u32)

EMB_OP(float, uint8_t, emb_u8_f32)
EMB_OP(double, uint8_t, emb_u8_f64)
EMB_OP(uint8_t, uint8_t, emb_u8_u8)
EMB_OP(uint32_t, uint8_t, emb_u8_u32)

IS_OP(float, uint32_t, is_u32_f32)
IS_OP(double, uint32_t, is_u32_f64)
IS_OP(uint8_t, uint32_t, is_u32_u8)
IS_OP(uint32_t, uint32_t, is_u32_u32)

IS_OP(float, uint8_t, is_u8_f32)
IS_OP(double, uint8_t, is_u8_f64)
IS_OP(uint8_t, uint8_t, is_u8_u8)
IS_OP(uint32_t, uint8_t, is_u8_u32)

GATHER_OP(float, uint32_t, gather_u32_f32)
GATHER_OP(double, uint32_t, gather_u32_f64)
GATHER_OP(uint8_t, uint32_t, gather_u32_u8)
GATHER_OP(uint32_t, uint32_t, gather_u32_u32)

GATHER_OP(float, uint8_t, gather_u8_f32)
GATHER_OP(double, uint8_t, gather_u8_f64)
GATHER_OP(uint8_t, uint8_t, gather_u8_u8)
GATHER_OP(uint32_t, uint8_t, gather_u8_u32)
