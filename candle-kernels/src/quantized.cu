// Kernels adapted from llama.cpp ggml-cuda.cu
// https://github.com/ggerganov/llama.cpp/blob/master/ggml-cuda.cu

#include <hip/hip_runtime.h>
#include "hip/hip_fp16.h"
#include "hip/hip_bf16.h"
#include<stdint.h>

typedef uint16_t ggml_fp16_t;

#define CUDA_USE_TENSOR_CORES

#define WARP_SIZE 32
#define CUDART_HMAX     11070 // CUDA 11.7, min. ver. for which __hmax and __hmax2 are known to work (may be higher than needed)

#define CC_PASCAL     600
#define MIN_CC_DP4A   610 // minimum compute capability for __dp4a, an intrinsic for byte-wise dot products
#define CC_VOLTA      700
#define CC_OFFSET_AMD 1000000
#define CC_RDNA1      (CC_OFFSET_AMD + 1010)
#define CC_RDNA2      (CC_OFFSET_AMD + 1030)
#define CC_RDNA3      (CC_OFFSET_AMD + 1100)

#define  MMQ_X_Q4_0_RDNA2  64
#define  MMQ_Y_Q4_0_RDNA2  128
#define NWARPS_Q4_0_RDNA2  8
#define  MMQ_X_Q4_0_RDNA1  64
#define  MMQ_Y_Q4_0_RDNA1  64
#define NWARPS_Q4_0_RDNA1  8
#if defined(CUDA_USE_TENSOR_CORES)
#define  MMQ_X_Q4_0_AMPERE 4
#define  MMQ_Y_Q4_0_AMPERE 32
#define NWARPS_Q4_0_AMPERE 4
#else
#define  MMQ_X_Q4_0_AMPERE 64
#define  MMQ_Y_Q4_0_AMPERE 128
#define NWARPS_Q4_0_AMPERE 4
#endif
#define  MMQ_X_Q4_0_PASCAL 64
#define  MMQ_Y_Q4_0_PASCAL 64
#define NWARPS_Q4_0_PASCAL 8

// QK = number of values after dequantization
// QR = QK / number of values before dequantization
// QI = number of 32 bit integers before dequantization

#define QK4_0 32
#define QR4_0 2
#define QI4_0 (QK4_0 / (4 * QR4_0))
typedef struct {
    half    d;              // delta
    uint8_t qs[QK4_0 / 2];  // nibbles / quants
} block_q4_0;
static_assert(sizeof(block_q4_0) == sizeof(ggml_fp16_t) + QK4_0 / 2, "wrong q4_0 block size/padding");

#define QK4_1 32
#define QR4_1 2
#define QI4_1 (QK4_1 / (4 * QR4_1))
typedef struct {
    half2   dm;             // dm.x = delta, dm.y = min
    uint8_t qs[QK4_1 / 2];  // nibbles / quants
} block_q4_1;
static_assert(sizeof(block_q4_1) == sizeof(ggml_fp16_t) * 2 + QK4_1 / 2, "wrong q4_1 block size/padding");

#define QK5_0 32
#define QR5_0 2
#define QI5_0 (QK5_0 / (4 * QR5_0))
typedef struct {
    half d;                 // delta
    uint8_t qh[4];          // 5-th bit of quants
    uint8_t qs[QK5_0 / 2];  // nibbles / quants
} block_q5_0;
static_assert(sizeof(block_q5_0) == sizeof(ggml_fp16_t) + sizeof(uint32_t) + QK5_0 / 2, "wrong q5_0 block size/padding");

#define QK5_1 32
#define QR5_1 2
#define QI5_1 (QK5_1 / (4 * QR5_1))
typedef struct {
    half2 dm;               // dm.x = delta, dm.y = min
    uint8_t qh[4];          // 5-th bit of quants
    uint8_t qs[QK5_1 / 2];  // nibbles / quants
} block_q5_1;
static_assert(sizeof(block_q5_1) == 2 * sizeof(ggml_fp16_t) + sizeof(uint32_t) + QK5_1 / 2, "wrong q5_1 block size/padding");

#define QK8_0 32
#define QR8_0 1
#define QI8_0 (QK8_0 / (4 * QR8_0))
typedef struct {
    half    d;              // delta
    int8_t  qs[QK8_0];      // quants
} block_q8_0;
static_assert(sizeof(block_q8_0) == sizeof(ggml_fp16_t) + QK8_0, "wrong q8_0 block size/padding");

#define QK8_1 32
#define QR8_1 1
#define QI8_1 (QK8_1 / (4 * QR8_1))
typedef struct {
    half2   ds;             // ds.x = delta, ds.y = sum
    int8_t  qs[QK8_0];      // quants
} block_q8_1;
static_assert(sizeof(block_q8_1) == 2*sizeof(ggml_fp16_t) + QK8_0, "wrong q8_1 block size/padding");

typedef float (*vec_dot_q_cuda_t)(const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & iqs);
typedef void (*allocate_tiles_cuda_t)(int ** x_ql, half2 ** x_dm, int ** x_qh, int ** x_sc);
typedef void (*load_tiles_cuda_t)(
    const void * __restrict__ vx, int * __restrict__ x_ql, half2 * __restrict__ x_dm, int * __restrict__ x_qh,
    int * __restrict__ x_sc, const int & i_offset, const int & i_max, const int & k, const int & blocks_per_row);
typedef float (*vec_dot_q_mul_mat_cuda_t)(
    const int * __restrict__ x_ql, const half2 * __restrict__ x_dm, const int * __restrict__ x_qh, const int * __restrict__ x_sc,
    const int * __restrict__ y_qs, const half2 * __restrict__ y_ms, const int & i, const int & j, const int & k);

// VDR = vec dot ratio, how many contiguous integers each thread processes when the vec dot kernel is called
// MMVQ = mul_mat_vec_q, MMQ = mul_mat_q

#define VDR_Q4_0_Q8_1_MMVQ 2
#define VDR_Q4_0_Q8_1_MMQ  4

template <int vdr> static __device__ __forceinline__ float vec_dot_q4_0_q8_1_impl(
    const int * v, const int * u, const float & d4, const half2 & ds8) {

    int sumi = 0;

#pragma unroll
    for (int i = 0; i < vdr; ++i) {
        const int vi0 = (v[i] >> 0) & 0x0F0F0F0F;
        const int vi1 = (v[i] >> 4) & 0x0F0F0F0F;

        // SIMD dot product of quantized values
        sumi = __dp4a(vi0, u[2*i+0], sumi);
        sumi = __dp4a(vi1, u[2*i+1], sumi);
    }

    const float2 ds8f = __half22float2(ds8);

    // second part effectively subtracts 8 from each quant value
    return d4 * (sumi * ds8f.x - (8*vdr/QI4_0) * ds8f.y);
}


static __device__ __forceinline__ float vec_dot_q4_0_q8_1_mul_mat(
    const int * __restrict__ x_ql, const half2 * __restrict__ x_dm, const int * __restrict__ x_qh, const int * __restrict__ x_sc,
    const int * __restrict__ y_qs, const half2 * __restrict__ y_ds, const int & i, const int & j, const int & k) {
    (void)x_qh; (void)x_sc;

    const int kyqs = k % (QI8_1/2) + QI8_1 * (k / (QI8_1/2));
    const float * x_dmf = (const float *) x_dm;

    int u[2*VDR_Q4_0_Q8_1_MMQ];

#pragma unroll
    for (int l = 0; l < VDR_Q4_0_Q8_1_MMQ; ++l) {
        u[2*l+0] = y_qs[j * WARP_SIZE + (kyqs + l)         % WARP_SIZE];
        u[2*l+1] = y_qs[j * WARP_SIZE + (kyqs + l + QI4_0) % WARP_SIZE];
    }

    return vec_dot_q4_0_q8_1_impl<VDR_Q4_0_Q8_1_MMQ>
        (&x_ql[i * (WARP_SIZE + 1) + k], u, x_dmf[i * (WARP_SIZE/QI4_0) + i/QI4_0 + k/QI4_0],
         y_ds[j * (WARP_SIZE/QI8_1) + (2*k/QI8_1) % (WARP_SIZE/QI8_1)]);
}
