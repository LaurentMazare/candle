#include "hip/hip_runtime.h"
// TODO: Use a proper distributed reduction rather than atomicAdd.
// https://people.maths.ox.ac.uk/gilesm/cuda/prac4/reduction.pdf
#include "cuda_utils.cuh"
#include<stdint.h>

const int BLOCK_SIZE = 1024;

// TODO: Maybe add some fast_sum_f16_f32 variant that not only accumulate in f32 but
// also expect a f32 output so that this can be used for normalization e.g. in softmax.

// Fast reduce sum kernel, this assumes that the dimensions to loop over are at
// the end, each block is responsible for populating one value in the output array.
// There are at most 1024 threads per block.
template <typename T>
__device__ void fast_sum(
    const size_t src_numel,
    const size_t el_to_sum_per_block,
    const size_t num_dims, 
    const size_t *info,
    const T *src,
    T *dst
) {
  const size_t *dims = info;
  const size_t *strides = info + num_dims;

  __shared__ T shr[BLOCK_SIZE];
  size_t tid = threadIdx.x;
  size_t dst_id = blockIdx.x;

  shr[tid] = 0.0;
  // Elements summed in this block range from dst_id * el_to_sum_per_block
  // to (dst_id + 1) * el_to_sum_per_block.
  size_t start_idx = dst_id * el_to_sum_per_block;
  size_t stop_idx = min(start_idx + el_to_sum_per_block, src_numel);
  size_t idx = start_idx + tid;

  while (idx < stop_idx) {
    // TODO: Fast version for the contiguous case.
    size_t strided_i = get_strided_index(idx, num_dims, dims, strides);
    shr[tid] += src[strided_i];
    idx += blockDim.x;
  }

  // Parallel reduction, see the slides:
  // https://www.olcf.ornl.gov/wp-content/uploads/2019/12/05_Atomics_Reductions_Warp_Shuffle.pdf
  // https://stackoverflow.com/questions/66078814/is-cuda-atomicadd-operation-faster-than-launch-another-kernel-when-we-do-reduce
  for (int s = blockDim.x / 2; s > 0; s >>= 1) {
    __syncthreads();
    if (tid < s) shr[tid] += shr[tid + s];
  }

  if (tid == 0) atomicAdd(dst + dst_id, shr[0]);
}

#define FAST_SUM_OP(TYPENAME, FN_NAME) \
extern "C" __global__ void FN_NAME(  \
    const size_t src_numel, \
    const size_t el_to_sum_per_block, \
    const size_t num_dims,  \
    const size_t *info, \
    const TYPENAME *src, \
    TYPENAME *dst \
) {  \
  fast_sum(src_numel, el_to_sum_per_block, num_dims, info, src, dst); \
} \

#define SUM_OP(TYPENAME, FN_NAME) \
extern "C" __global__ void FN_NAME(  \
    const size_t numel,  \
    const size_t num_dims, \
    const size_t num_sum_dims, \
    const size_t *info, \
    const TYPENAME *inp, \
    TYPENAME *out \
) {  \
    const size_t *dims = info; \
    const size_t *strides = info + num_dims; \
    const size_t *sum_dims_l = info + 2*num_dims; \
    const size_t *sum_dims_s = info + 2*num_dims + num_sum_dims; \
    if (is_contiguous(num_dims, dims, strides)) { \
        for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) { \
            size_t dst_index = i; \
            for (unsigned int nd = 0; nd < num_sum_dims; ++nd) { \
              size_t stride = sum_dims_s[nd]; \
              size_t pre = dst_index / stride; \
              size_t post = dst_index % stride; \
              dst_index = (pre / sum_dims_l[nd]) * stride + post; \
            } \
            atomicAdd(out + dst_index, inp[i]); \
        } \
    } \
    else { \
        for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) { \
            unsigned strided_i = get_strided_index(i, num_dims, dims, strides); \
            size_t dst_index = i; \
            for (unsigned int nd = 0; nd < num_sum_dims; ++nd) { \
              size_t stride = sum_dims_s[nd]; \
              size_t pre = dst_index / stride; \
              size_t post = dst_index % stride; \
              dst_index = (pre / sum_dims_l[nd]) * stride + post; \
            } \
            atomicAdd(out + dst_index, inp[strided_i]); \
        } \
    } \
} \

#if __CUDA_ARCH__ >= 800
SUM_OP(__hip_bfloat16, sum_bf16)
FAST_SUM_OP(__hip_bfloat16, fast_sum_bf16)
#endif

#if __CUDA_ARCH__ >= 530
SUM_OP(__half, sum_f16)
FAST_SUM_OP(__half, fast_sum_f16)
#endif

SUM_OP(float, sum_f32)
SUM_OP(double, sum_f64)
SUM_OP(uint32_t, sum_u32)

FAST_SUM_OP(float, fast_sum_f32)
FAST_SUM_OP(double, fast_sum_f64)
FAST_SUM_OP(uint32_t, fast_sum_u32)
