#include "hip/hip_runtime.h"
// WARNING: THIS IS ONLY VALID ASSUMING THAT inp IS CONTIGUOUS!
// TODO: proper error reporting when ids are larger than v_size.
#include "cuda_utils.cuh"
#include<stdint.h>

#define EMB_OP(TYPENAME, FN_NAME) \
extern "C" __global__ void FN_NAME(  \
    const size_t numel,  \
    const size_t num_dims, \
    const size_t *info, \
    const uint32_t *ids, \
    const TYPENAME *inp, \
    TYPENAME *out, \
    const size_t h_size, \
    const size_t v_size \
) {  \
    const size_t *dims = info; \
    const size_t *strides = info + num_dims; \
    if (is_contiguous(num_dims, dims, strides)) { \
        for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) { \
            memcpy(&out[i * h_size], &inp[ids[i] * h_size], h_size * sizeof(TYPENAME)); \
        } \
    } \
    else { \
        for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) { \
            unsigned strided_i = get_strided_index(i, num_dims, dims, strides); \
            memcpy(&out[i * h_size], &inp[ids[strided_i] * h_size], h_size * sizeof(TYPENAME)); \
        } \
    } \
} \

#if __CUDA_ARCH__ >= 530
EMB_OP(__half, emb_f16)
#endif

EMB_OP(float, emb_f32)
EMB_OP(double, emb_f64)
EMB_OP(uint32_t, emb_u32)
