#include "hip/hip_runtime.h"
#include "binary_op_macros.cuh"
#include<stdint.h>

#if __CUDA_ARCH__ >= 530
BINARY_OP(__half, badd_f16, x + y)
BINARY_OP(__half, bdiv_f16, x / y)
BINARY_OP(__half, bmul_f16, x * y)
BINARY_OP(__half, bsub_f16, x - y)
#endif

BINARY_OP(float, badd_f32, x + y)
BINARY_OP(double, badd_f64, x + y);
BINARY_OP(uint32_t, badd_u32, x + y);
BINARY_OP(float, bdiv_f32, x / y)
BINARY_OP(double, bdiv_f64, x / y);
BINARY_OP(uint32_t, bdiv_u32, x / y);
BINARY_OP(float, bmul_f32, x * y)
BINARY_OP(double, bmul_f64, x * y);
BINARY_OP(uint32_t, bmul_u32, x * y);
BINARY_OP(float, bsub_f32, x - y)
BINARY_OP(double, bsub_f64, x - y);
BINARY_OP(uint32_t, bsub_u32, x - y);
