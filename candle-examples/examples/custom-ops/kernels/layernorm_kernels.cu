#include "hip/hip_runtime.h"
#include "reduction_utils.cuh"

template <typename scalar_t>
__device__ void
rms_norm_kernel(scalar_t *__restrict__ out,         // [num_tokens, hidden_size]
                const scalar_t *__restrict__ input, // [num_tokens, hidden_size]
                const scalar_t *__restrict__ weight, // [hidden_size]
                const float epsilon, const int num_tokens,
                const int hidden_size) {
  __shared__ float s_variance;
  float variance = 0.0f;

  for (int idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    const float x = (float)input[blockIdx.x * hidden_size + idx];
    variance += x * x;
  }
  variance = blockReduceSum<float>(variance);
  if (threadIdx.x == 0) {
    s_variance = rsqrtf(variance / hidden_size + epsilon);
  }
  __syncthreads();

  for (int idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    float x = (float)input[blockIdx.x * hidden_size + idx];
    out[blockIdx.x * hidden_size + idx] =
        ((scalar_t)(x * s_variance)) * weight[idx];
  }
}
extern "C" __global__ void rms_norm_kernel_f32(
    float *__restrict__ out,         // [num_tokens, hidden_size]
    const float *__restrict__ input, // [num_tokens, hidden_size]
    const float *__restrict__ weight, // [hidden_size]
    const float epsilon, const int num_tokens,
    const int hidden_size) {
  rms_norm_kernel(out, input, weight, epsilon, num_tokens, hidden_size);
}

